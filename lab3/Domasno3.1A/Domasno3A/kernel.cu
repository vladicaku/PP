#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <gputimer.h>
using namespace std;

const int arraySize = 1024;

__device__ int getGlobalIdx_3D_3D(){
	int blockId = blockIdx.x + blockIdx.y * gridDim.x
		+ gridDim.x * gridDim.y * blockIdx.z;
	int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
		+ (threadIdx.z * (blockDim.x * blockDim.y))
		+ (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__global__ void matrix_add(float *a, float *b, float *c)
{
	int globalId = getGlobalIdx_3D_3D();
	c[globalId] = a[globalId] + b[globalId];
}

int main()
{
	float *h_a, *h_b, *h_c;	
	float *d_a, *d_b, *d_c;
	int bytes = arraySize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);
	h_b = (float*) malloc(bytes);
	h_c = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);
	hipMalloc((void **) &d_c, bytes);

	// init host arrays
	for (int i=0; i<arraySize; i++) 
	{
		h_a[i] = i;
		h_b[i] = i;
		h_c[i] = 0;
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);
	hipMemset(d_c, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);


	// kernel call
	timer.Start();
	matrix_add<<<1, 1024>>>(d_a, d_b, d_c);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	// copy to host
	/*hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
	for (int i=0; i<arraySize; i++)
	{
		cout << " " << h_c[i];
	}*/


	scanf("%d", NULL);
	return 0;
	return 0;
}

