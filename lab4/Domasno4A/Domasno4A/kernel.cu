#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include <stdio.h>
#include <stdlib.h> 
#include <iostream>
using namespace std;

const int arraySize = 2048;

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ int getGlobalIdx_1D_1D() 
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void vector_add_global(float *a, float *b, float *c)
{
    int globalId = getGlobalIdx_1D_1D();
    c[globalId] = a[globalId] + b[globalId];
}

__global__ void vector_add_shared(float *a, float *b, float *c)
{
	extern __shared__ float sum[];
    int globalId = getGlobalIdx_1D_1D();
	int inBlockId = threadIdx.x + threadIdx.y * blockDim.x;
	
	sum[inBlockId] = a[globalId];
	sum[inBlockId] += b[globalId];
    c[globalId] = sum[inBlockId];
}

int main()
{
	float *h_a, *h_b, *h_c;	
	float *d_a, *d_b, *d_c;
	int bytes = arraySize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);
	h_b = (float*) malloc(bytes);
	h_c = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);
	hipMalloc((void **) &d_c, bytes);

	// init host arrays
	for (int i=0; i<arraySize; i++) 
	{
			h_a[i] = i;
			h_b[i] = i;
			h_c[i] = 0;
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);
	hipMemset(d_c, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
	
	
	// kernel call
	timer.Start();
	//vector_add_global<<<8, 256>>>(d_a, d_b, d_c);
	vector_add_shared<<<2, 1024, 1024*sizeof(float)>>>(d_a, d_b, d_c);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	timer.Start();
	vector_add_shared<<<4, 512, 512*sizeof(float)>>>(d_a, d_b, d_c);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	timer.Start();
	vector_add_shared<<<8, 256, 256*sizeof(float)>>>(d_a, d_b, d_c);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	// copy to host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
	for (int i=0; i<arraySize; i++)
	{
		cout << " " << h_c[i];
	}


	scanf("%d", NULL);
    return 0;
}

