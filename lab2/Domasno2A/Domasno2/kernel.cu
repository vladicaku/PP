#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include <stdio.h>
#include <stdlib.h> 
#include <iostream>
using namespace std;

const int arraySize = 1024;

__global__ void vector_add(float *a, float *b, float *c)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int id = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	//int id = threadIdx.x + threadIdx.y * blockDim.x;
    c[id] = a[id] + b[id];
}

int main()
{
    float *h_a, *h_b, *h_c;	
	float *d_a, *d_b, *d_c;
	int bytes = arraySize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);
	h_b = (float*) malloc(bytes);
	h_c = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);
	hipMalloc((void **) &d_c, bytes);

	// init host arrays
	for (int i=0; i<arraySize; i++) 
	{
			h_a[i] = i;
			h_b[i] = i;
			h_c[i] = 0;
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);
	hipMemset(d_c, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
	
	
	// kernel call
	timer.Start();
	vector_add<<<dim3(4, 1, 1), dim3(16, 16, 1)>>>(d_a, d_b, d_c);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	// copy to host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
	for (int i=0; i<arraySize; i++)
	{
		cout << " " << h_c[i];
	}


	scanf("%d", NULL);
    return 0;
}
