#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
using namespace std;

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ int getGlobalIdx_1D_1D() 
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void transpose_serial(float *a, float *b, int N)
{
	for (int j=0; j<N; j++)
	{
		for (int i=0; i<N; i++)
		{
			b[j+i*N] = a[i + j*N];
		}
	}
}

__global__ void transpose_row_paralel(float *a, float *b, int N)
{
	int i = threadIdx.x;
	for (int j=0; j<N; j++)
	{
		b[j+i*N] = a[i+j*N];
	}
}

__global__ void transpose_element_paralel(float *a, float *b, int N)
{
	// absolute (real) 2D location
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	b[j+i*N] = a[i+j*N];
}


void test(int g1, int g2, int b1, int b2, int kernel, int matrixSize, bool showOutput)
{
	float *h_a, *h_b;
	float *d_a, *d_b;
	int bytes = matrixSize * matrixSize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);
	h_b = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);

	// init host arrays
	srand(time(NULL));
	for (int i=0; i<matrixSize; i++) 
	{
		for (int j=0; j<matrixSize; j++)
		{
			h_a[i*matrixSize + j] = 10*i+j;
			h_b[i*matrixSize + j] = 0;
		}
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// kernel call
	if (kernel == 0)
	{
		timer.Start();
		transpose_serial<<<dim3(g1, g2), dim3(b1, b2)>>>(d_a, d_b, matrixSize);
		timer.Stop();
	}
	else if (kernel == 1)
	{	
		timer.Start();
		transpose_row_paralel<<<dim3(g1, g2), dim3(b1, b2)>>>(d_a, d_b, matrixSize);
		timer.Stop();
	}
	else if (kernel == 2)
	{
		timer.Start();
		transpose_element_paralel<<<dim3(g1, g2), dim3(b1, b2)>>>(d_a, d_b, matrixSize);	
		timer.Stop();
	}

	cout << timer.Elapsed() << "\t<<<" << g << ", " << b << ">>> " << "\n";

	// copy to host
	hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
	if (showOutput) 
	{
		for (int i=0; i<matrixSize; i++)
		{
			for (int j=0; j<matrixSize; j++)
			{
				cout << " " << h_b[i*matrixSize + j];
			}
			cout << endl;
		}
		cout << "\n";
	}
	hipFree(d_a);
	hipFree(d_b);
}


void test1(int g1, int g2, int b1, int b2, int matrixSize, bool showOutput)
{
	float *h_a, *d_a;
	float *h_b, *d_b;
	int bytes = matrixSize * matrixSize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);
	h_b = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);

	// init host arrays
	srand(time(NULL));
	for (int i=0; i<matrixSize; i++) 
	{
		for (int j=0; j<matrixSize; j++)
		{
			h_a[i*matrixSize + j] = 10*i+j;
			h_b[i*matrixSize + j] = 0;
		}
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// kernel call
	timer.Start();
	transpose_element_paralel<<<dim3(g1, g2), dim3(b1, b2)>>>(d_a, d_b, matrixSize);	
	timer.Stop();
	
	cout << timer.Elapsed() << "\t<<<dim3(" << g1 << ", " << g2 << "), dim3(" << b1 << ", " << b2 << ")>>>\n";

	// copy to host
	hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
	if (showOutput) 
	{
		for (int i=0; i<matrixSize; i++)
		{
			for (int j=0; j<matrixSize; j++)
			{
				cout << " " << h_b[i*matrixSize + j];
			}
			cout << endl;
		}
		cout << "\n";
	}
	hipFree(d_a);
	hipFree(d_b);
}


int main()
{
	cout << "Matrix size: 10x10" << "\n";
	test(1, 1, 0, 10, true);
	cout << "\n";
	test(1, 10, 1, 10, true);
	cout << "\n";
	test1(5, 5, 2, 2, 10, true); // different test procedure
	cout << "\n";

	cout << "\nMatrix size: 40x40" << "\n";
	test(1, 1, 0, 40, false);
	test(1, 40, 1, 40, false);
	test1(10, 10, 4, 4, 40, false);  // different test procedure
	cout << "\n";

	cout << "\nMatrix size: 100x100" << "\n";
	test(1, 1, 0, 100, false);
	test(1, 100, 1, 100, false);
	test1(25, 25, 4, 4, 100, false);  // different test procedure
	cout << "\n";

    cout << "\nMatrix size: 1024x1024" << "\n";
	test(1, 1, 0, 1024, false);
	test(1, 1024, 1, 1024, false);
	test1(128, 128, 8, 8, 1024, false);  // different test procedure
	test1(32, 32, 32, 32, 1024, false);  // different test procedure
	cout << "\n";


	
	
	scanf("%d", NULL);
    return 0;

}

