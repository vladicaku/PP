#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <gputimer.h>
using namespace std;

const int matrixSize = 1024;

__device__ int getGlobalIdx_3D_3D(){
	int blockId = blockIdx.x + blockIdx.y * gridDim.x
		+ gridDim.x * gridDim.y * blockIdx.z;
	int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
		+ (threadIdx.z * (blockDim.x * blockDim.y))
		+ (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__global__ void matrix_add(float *a, float *b, float *c)
{
	int globalId = getGlobalIdx_3D_3D();
	c[globalId] = a[globalId] + b[globalId];
}

int main()
{
	float *h_a, *h_b, *h_c;	
	float *d_a, *d_b, *d_c;
	int bytes = matrixSize * matrixSize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);
	h_b = (float*) malloc(bytes);
	h_c = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);
	hipMalloc((void **) &d_c, bytes);

	// init host arrays
	for (int i=0; i<matrixSize*matrixSize; i++) 
	{
			h_a[i] = i;
			h_b[i] = i;
			h_c[i] = 0;
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);
	hipMemset(d_c, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);


	// kernel call
	timer.Start();
	matrix_add<<<dim3(2,128,4), dim3(2,128,4)>>>(d_a, d_b, d_c);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	// copy to host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
	/*for (int i=0; i<matrixSize; i++)
	{
		for (int j=0; j<matrixSize; j++)
		{
		cout << " " << h_c[i*matrixSize + j];
		}
		cout << endl;
	}*/
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	scanf("%d", NULL);
    return 0;
}

