#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void rgba_to_greyscale_strided(const uchar4* const rgbaImage, unsigned char* const greyImage, int numRows, int numCols)
{
	int abs_x = blockIdx.x * blockDim.x + threadIdx.x;
	int abs_y = blockIdx.y * blockDim.y + threadIdx.y;
	int globalId;

	if (abs_x <= numCols && abs_y <= numRows)
	{
		globalId = abs_x + abs_y * numCols;
		greyImage[globalId] = .299f * rgbaImage[globalId].x + .587f * rgbaImage[globalId].y + .114f * rgbaImage[globalId].z;
		//rgba_to_greyscale(rgbaImage, greyImage, numRows, numCols);
	}
}
// 0.033376 ms 


__device__ int getGlobalIdx_2D_2D() 
{
		int blockId = blockIdx.x + blockIdx.y * gridDim.x;
		int threadId = blockId * (blockDim.x * blockDim.y)
			+ (threadIdx.y * blockDim.x) + threadIdx.x;
		return threadId;
}


__global__ void rgba_to_greyscale_naive_coalesced(const uchar4* const rgbaImage, unsigned char* const greyImage, int len)
{
	int globalId = getGlobalIdx_2D_2D();

	if (globalId < len) {
		greyImage[globalId] = .299f * rgbaImage[globalId].x + .587f * rgbaImage[globalId].y + .114f * rgbaImage[globalId].z;
	}
}
/*
0.029856 ms 
const dim3 blockSize(20, 20, 1);  //TODO
const dim3 gridSize(numCols/20 + 1, numRows/20 + 1, 1);  //TODO
*/

__global__ void rgba_to_greyscale_noif_coalesced(const uchar4* const rgbaImage, unsigned char* const greyImage)
{
	int globalId = getGlobalIdx_2D_2D();
	greyImage[globalId] = .299f * rgbaImage[globalId].x + .587f * rgbaImage[globalId].y + .114f * rgbaImage[globalId].z;
}

int main()
{
	return 0;
}

