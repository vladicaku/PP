#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
using namespace std;

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ int getGlobalIdx_1D_1D() 
{
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void dot_product(float *a, float *b, float *total, int N)
{
	extern __shared__  float temp[];
	int threadId = getGlobalIdx_1D_1D();

	if (threadId < N) 
	{
		temp[threadIdx.x] = a[threadId] * b[threadId];
	}

	__syncthreads();

	// reduction
	int i  = blockDim.x / 2 ;
	while (i != 0)
	{
		if (threadIdx.x < i)
		{
			temp[threadIdx.x] += temp[threadIdx.x + i];
		}
		__syncthreads();
		i /= 2;
	}

	//__syncthreads();
	

	// atomic add sum from each block
	if (threadIdx.x == 0)
	{
		atomicAdd(total, temp[threadIdx.x]);
	}
}

void test(int g, int b, int kernel, int arraySize, bool showOutput)
{
	float *h_a, *h_b, *h_total;
	float *d_a, *d_b, *d_total;
	int bytes = arraySize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);
	h_b = (float*) malloc(bytes);
	h_total = (float*) malloc(sizeof(float));

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);
	hipMalloc((void **) &d_total, sizeof(float));

	// init host arrays
	for (int i=0; i<arraySize; i++) 
	{
		h_a[i] = i+1;
		h_b[i] = i+1;
	}
	h_total[0] = 0;

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);
	hipMemset(d_total, 0, sizeof(float));

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_total, h_total, sizeof(float), hipMemcpyHostToDevice);

	// kernel call
	if (kernel == 0)
	{
		timer.Start();
		dot_product<<<g, b, b*sizeof(float)>>>(d_a, d_b, d_total, arraySize);
		timer.Stop();
	}
	
	cout << timer.Elapsed() << "\t<<<" << g << ", " << b << ">>> " << "\n";

	// copy to host
	hipMemcpy(h_total, d_total, sizeof(float), hipMemcpyDeviceToHost);

	// show output
	if (showOutput) 
	{
		cout << h_total[0] << "\n";
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_total);
}


int main()
{
	// g, b, kernel, arraySize, showOutput
	cout << "Array size 2048" << "\n" << "\n";
	test(2, 1024, 0, 2048, true); 
	cout << "\n";
	test(4, 512, 0, 2048, true); 
	cout << "\n";
	test(8, 256, 0, 2048, true); 
	cout << "\n";
	test(16, 128, 0, 2048, true); 
	cout << "\n";
	test(32, 64, 0, 2048, true); 
	cout << "\n";

	cout << "\n";
	cout << "\n";
	cout << "Array size 16384" << "\n" << "\n";
	test(16, 1024, 0, 16384, true); 
	cout << "\n";
	test(32, 512, 0, 16384, true); 
	cout << "\n";
	test(64, 256, 0, 16384, true); 
	cout << "\n";
	test(128, 128, 0, 16384, true); 
	cout << "\n";

	scanf("%d", NULL);
    return 0;
}

