#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    return 0;
}

