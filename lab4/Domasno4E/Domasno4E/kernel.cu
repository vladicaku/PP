#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
using namespace std;

const int arraySize = 1024;

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ int getGlobalIdx_1D_1D() 
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void find_max(int *a, int *b)
{
	int globalId = getGlobalIdx_2D_2D();
	atomicMax(b, a[globalId]);
}



void test1(int g, int b)
{
	int *h_a, *h_b;
	int *d_a, *d_b;
	int bytes = arraySize * sizeof(int);
	GpuTimer timer;

	h_a = (int*) malloc(bytes);
	h_b = (int*) malloc(sizeof(int));

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, sizeof(int));

	// init host arrays
	for (int i=0; i<arraySize; i++) 
	{
		h_a[i] = i;
	}
	h_b[0] = 0;

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, sizeof(int));

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(int), hipMemcpyHostToDevice);


	// kernel call
	timer.Start();
	find_max<<<g, b>>>(d_a, d_b);
	timer.Stop();
	cout << timer.Elapsed() << "\t<<<" << g << ", " << b << ">>> " << "\n";

	// copy to host
	hipMemcpy(h_b, d_b, sizeof(int), hipMemcpyDeviceToHost);
	for (int i=0; i<1; i++)
	{
		cout << h_b[i] << " ";
	}	
	cout << "\n";
}

int main()
{
	test1(1, 1024);
	test1(2, 512);
	test1(4, 256);
	test1(8, 128);
	test1(512, 2);
	scanf("%d", NULL);
	return 0;
}
