#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <time.h> // for cpu 
using namespace std;

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ int getGlobalIdx_1D_1D() 
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void transpose_serial(float *a, float *b, int N)
{
	for (int j=0; j<N; j++)
	{
		for (int i=0; i<N; i++)
		{
			b[j+i*N] = a[i + j*N];
		}
	}
}

__global__ void transpose_row_paralel(float *a, float *b, int N)
{
	int i = threadIdx.x;
	for (int j=0; j<N; j++)
	{
		b[j+i*N] = a[i+j*N];
	}
}

__global__ void transpose_element_paralel(float *a, float *b, int N)
{
	// absolute (real) 2D location, invalid code, works only for 1 block
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;


	//b[j + i * N] = a[i + j * N];
	// small fix
	if (i <= N && j <= N)
	{
		b[j + i * N] = a[i + j * N];
	}
	
}

void transpose_cpu(float *a, float *b, int N)
{
	for (int j=0; j<N; j++)
	{
		for (int i=0; i<N; i++)
		{
			b[j+i*N] = a[i + j*N];
		}
	}
}

void test(int g1, int g2, int b1, int b2, int kernel, int matrixSize, bool showOutput)
{
	float *h_a, *h_b;
	float *d_a, *d_b;
	int bytes = matrixSize * matrixSize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);
	h_b = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);

	// init host arrays
	srand(time(NULL));
	for (int i=0; i<matrixSize; i++) 
	{
		for (int j=0; j<matrixSize; j++)
		{
			h_a[i*matrixSize + j] = 10*i+j+1;
			h_b[i*matrixSize + j] = 0;
		}
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// kernel call
	if (kernel == 0)
	{
		timer.Start();
		transpose_serial<<<dim3(g1, g2), dim3(b1, b2)>>>(d_a, d_b, matrixSize);
		timer.Stop();
		cout << "CUDA serial" << "\n";
	}
	else if (kernel == 1)
	{	
		timer.Start();
		transpose_row_paralel<<<dim3(g1, g2), dim3(b1, b2)>>>(d_a, d_b, matrixSize);
		timer.Stop();
		cout << "CUDA row parallel" << "\n";
	}
	else if (kernel == 2)
	{
		timer.Start();
		transpose_element_paralel<<<dim3(g1, g2), dim3(b1, b2)>>>(d_a, d_b, matrixSize);	
		timer.Stop();
		cout << "CUDA element parallel" << "\n";
	}
	else if (kernel == 3)
	{
		clock_t begin, end;
		double time_spent;

		/*
		// gettimeofday() isn't guaranteed to be monotonic
		timeval tm1;
		timeval tm2;
		gettimeofday(&tm1, NULL);
		transpose_cpu(h_a, h_b, matrixSize);
		gettimeofday(&tm2, NULL);
		unsigned long long time_spent = 1000 * (tm2.tv_sec - tm1.tv_sec) + (tm2.tv_usec - tm1.tv_usec) / 1000;
		printf("%llu ms\n", t);
		*/

		
		begin = clock();
		transpose_cpu(h_a, h_b, matrixSize);
		end = clock();
		//time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		//time_spent = (double)(end - begin);
		time_spent = ((double) (end - begin) * 1000.0) / CLOCKS_PER_SEC;
		

		cout << "CPU serial" << "\n";
		//printf("Time elapsed in ms: %f \n", time_spent);
		cout << time_spent << "\n";

		// show output
		if (showOutput) 
		{
			for (int i=0; i<matrixSize; i++)
			{
				for (int j=0; j<matrixSize; j++)
				{
					cout << " " << h_b[i*matrixSize + j];
				}
				cout << endl;
			}
			cout << "\n";
		}

		//cout << h_b[0] << " " << h_b[matrixSize * matrixSize - 1] << "\n";
		return;

	}

	cout << timer.Elapsed() << "\t<<<dim3(" << g1 << ", " << g2 << "), dim3(" << b1 << ", " << b2 << ")>>>\n";

	// copy to host
	hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);

	//cout << h_b[0] << " " << h_b[matrixSize * matrixSize - 1] << "\n";;

	// show output
	if (showOutput) 
	{
		for (int i=0; i<matrixSize; i++)
		{
			for (int j=0; j<matrixSize; j++)
			{
				cout << " " << h_b[i*matrixSize + j];
			}
			cout << endl;
		}
		cout << "\n";
	}

	hipFree(d_a);
	hipFree(d_b);
}


int main()
{
	// g1, g2, b1, b2, kernel, matrixSize, showOutput
	cout << "--------------------- Basic testing ---------------------" << "\n";
	cout << "Matrix size: 4x4" << "\n" << "\n";
	test(1, 1, 1, 1, 0, 4, true); // cuda serial
	cout << "\n";
	test(1, 1, 4, 1, 1, 4, true); // cuda row paralel
	cout << "\n";
	test(2, 2, 2, 2, 2, 4, true); // cuda element paralel
	cout << "\n";
	test(1, 1, 1, 1, 3, 4, true); // CPU serial

	cout << "\n";
	cout << "Matrix size: 40x40" << "\n" << "\n";
	test(1, 1, 1, 1, 0, 40, false); // cuda serial
	cout << "\n";
	test(1, 1, 40, 1, 1, 40, false); // cuda row paralel
	cout << "\n";
	test(4, 4, 10, 10, 2, 40, false); // cuda element paralel
	cout << "\n";
	test(1, 1, 1, 1, 3, 40, false); // CPU serial


	// Best grid and block size testing
	cout << "\n";
	cout << "--------------------- Grid and block size testing ---------------------" << "\n";
	cout << "\n";
	cout << "Matrix size: 100x100" << "\n" << "\n";
	test(4, 4, 25, 25, 2, 40, false); // cuda element paralel
	cout << "\n";
	test(5, 5, 20, 20, 2, 40, false); // cuda element paralel
	cout << "\n";
	test(10, 10, 10, 10, 2, 40, false); // cuda element paralel
	cout << "\n";
	test(20, 20, 5, 5, 2, 40, false); // cuda element paralel


	// Extreme testing
	// Tested 
	cout << "\n";
	cout << "--------------------- Extreme ---------------------" << "\n";
	cout << "\n";
	cout << "Matrix size: 500x500" << "\n" << "\n";
	test(50, 50, 10, 10, 2, 500, false); // cuda element paralel
	cout << "\n";
	test(1, 1, 1, 1, 3, 500, false); // CPU serial
	
	cout << "\n";
	cout << "Matrix size: 1000x1000" << "\n" << "\n";
	test(50, 50, 20, 20, 2, 1000, false); // cuda element paralel
	cout << "\n";
	test(1, 1, 1, 1, 3, 1000, false); // CPU serial

	
	scanf("%d", NULL);
    return 0;
}

