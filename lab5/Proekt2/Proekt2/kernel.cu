#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <math.h>	// ceil
#include "utils.h"	//hipError_t
using namespace std;

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ int getGlobalIdx_1D_1D() 
{
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
	int abs_x = blockIdx.x * blockDim.x + threadIdx.x;
	int abs_y = blockIdx.y * blockDim.y + threadIdx.y;
	float result = 0.f;

	if (abs_x < numCols && abs_y < numRows)
	{
		for (int filter_r = -filterWidth/2; filter_r <= filterWidth/2; ++filter_r) 
		{
			for (int filter_c = -filterWidth/2; filter_c <= filterWidth/2; ++filter_c) 
			{	
				int image_r = min(max(abs_y + filter_r, 0), static_cast<int>(numRows - 1));
				int image_c = min(max(abs_x + filter_c, 0), static_cast<int>(numCols - 1));
				
				float image_value = static_cast<float>(inputChannel[image_r * numCols + image_c]);
				float filter_value = filter[(filter_r + filterWidth/2) * filterWidth + filter_c + filterWidth/2];

				result += image_value * filter_value;
			}
		}

		 outputChannel[abs_y  * numCols + abs_x] = result;
	}
}

__global__ 
void separateChannels(const uchar4* const inputImageRGBA, int numRows, int numCols, 
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
	// posledovatelen memoriski pristap, namesto apsolutna pozicija 
	int threadId = getGlobalIdx_2D_2D();
	if (threadId < numRows * numCols)
	{
		redChannel[threadId] = inputImageRGBA[threadId].x;
		greenChannel[threadId] = inputImageRGBA[threadId].y;
		blueChannel[threadId] = inputImageRGBA[threadId].z;
	}
}

__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  checkCudaErrors(hipMalloc(&d_filter,  sizeof(float) * filterWidth * filterWidth));
  checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));
}

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
  //Set reasonable block size (i.e., number of threads per block)
  const dim3 blockSize(10, 10, 1);

  //Compute correct grid size (i.e., number of blocks per kernel launch)
  //from the image size and and block size.
  size_t gridX = ceil(numCols * 1.0 / blockSize.x);
  size_t gridY = ceil(numRows * 1.0 / blockSize.y);
  const dim3 gridSize(gridX, gridY, 1);

  //cout << "<<<dim3(" << gridSize.x << ", " << gridSize.y << "), dim3(" << blockSize.x << ", " << blockSize.y << "_>>>" << "\n";

  //Launch a kernel for separating the RGBA image into different color channels
  separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);

  // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  //Call your convolution kernel here 3 times, once for each color channel.
  gaussian_blur<<<gridSize, blockSize>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
  gaussian_blur<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
  gaussian_blur<<<gridSize, blockSize>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);

  // Again, call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Now we recombine your results. We take care of launching this kernel for you.
  //
  // NOTE: This kernel launch depends on the gridSize and blockSize variables,
  // which you must set yourself.
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}


void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}

int main()
{
}