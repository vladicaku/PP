#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h" // custom lib
#include <stdio.h>
#include <stdlib.h> 
#include <iostream>
#include <math.h>
using namespace std;

// Will be used to determinate the total number of blocks for the kernel. 
// The number of blocks is going to be		totalNumberOfBlocks = nVertices / NUMBER_OF_THREADS_PER_BLOCK.
// If nVertices % NUMBER_OF_THREADS_PER_BLOCK > 0, then one extra block will be lunched,
// i.e totalNumberOfBlocks will be increased by 1, totalNumberOfBlocks++.
const int NUMBER_OF_THREADS_PER_BLOCK = 128;

// Indicates that there are changes in the depths array (new depth is added in the array).
// If there are no changes, the loop in the main() will stop. This means that all 
// of the nodes in the graph are visited.
__device__ int d_hasChanges = 0;

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ int getGlobalIdx_1D_1D() 
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void bfs(int *vertices, int *edges, int *depths, int *frontiers, int n) {
	int threadId = getGlobalIdx_2D_2D();
	int numberOfEdges;
	int firstEdgePosition;

	//printf("TID: %d\n", threadId);

	if ((threadId < n) && (frontiers[threadId] == 1)) {
		frontiers[threadId] = 0;

		numberOfEdges = vertices[threadId + 1] - vertices[threadId];
		firstEdgePosition = vertices[threadId];

		//printf("TID: %d - numberOfEdges = %d\n", threadId, numberOfEdges);
		//printf("TID: %d - firstEdgePosition = %d\n", threadId, firstEdgePosition);
		int myDepht = depths[threadId];

		for (int i=0; i<numberOfEdges; i++) {
			int edgePosition = firstEdgePosition + i;
			int nodePosition = edges[edgePosition];

			//printf("TID: %d - nodePosition: %d; level: %d\n", threadId, nodePosition, level);
			//printf("TID: %d - depth[%d] = %d\n", threadId, nodePosition, depths[nodePosition]);
			if (depths[nodePosition] == -1) {
				depths[nodePosition] = myDepht + 1;
				frontiers[nodePosition]  = 1;
				d_hasChanges = 1;
				//printf("TID: %d - hasChanges: true; nodePosition: %d;  level: %d\n", threadId, nodePosition, level);
			}
		}
		
	}
}

// Generates a graph in form of rings.
// Each vertex has equal number of edges.
// File format:
// #ofVertices
// #ofEdges
// #ofEdges for current node
// currentNode endNode
// .
// .
// #ofEdges for current node
// currentNode endNode
// currentNode endNode
// .
// .
void generate(int level, int numberOfEdgesPerVertex) {
	printf("Generating input file ....\n");
	FILE *fp;
	fp = fopen("D:\\input.txt", "w+");
	int counter = 1;

	// calculate the total number of edges
	int totalNumberOfEdges = 0;
	for (int i=1; i<=level; i++) {
		totalNumberOfEdges += (int)(pow(numberOfEdgesPerVertex, i));
	}

	// write number of vertices
	fprintf(fp, "%d\n", totalNumberOfEdges + 1);
	// write number of edges
	fprintf(fp, "%d\n", totalNumberOfEdges);

	int currentVertex = -1;
	for (int i=0; i<totalNumberOfEdges; i++) {
		if (i % numberOfEdgesPerVertex == 0) {
			currentVertex++;
			fprintf(fp, "%d\n", numberOfEdgesPerVertex);
		}
		fprintf(fp, "%d %d\n", currentVertex, i+1);
	}

	for (int i=0; i<(int)(pow(numberOfEdgesPerVertex, level)); i++) {
		fprintf(fp,"0\n");
	}

	fclose(fp);
}

int main(int argc, char* argv[])
{
	// TODO
	// implement CLI arguments and print usage
	generate(2, 1000);
	GpuTimer timer;
	FILE *fp, *fp1;
	int nVertices = 0, nEdges = 0;
	int *h_vertices, *h_edges, *h_depths, *h_frontiers;
	int *d_vertices, *d_edges, *d_depths, *d_frontiers;

	// read from file
	fp = fopen("D:\\input.txt", "r");
	fscanf(fp, "%d", &nVertices);
	fscanf(fp, "%d", &nEdges);

	// set sizes
	int verticesSize = (nVertices + 1) * sizeof(int);
	int edgesSize = nEdges * sizeof(int);
	int depthsSize = nVertices * sizeof(int);
	int frontiersSize = nVertices * sizeof(int);

	// allocate host memory
	printf("Allocate host memory ...\n");
	h_vertices = (int*) malloc(verticesSize);
	h_edges = (int*) malloc(edgesSize);
	h_depths = (int*) malloc(depthsSize);
	h_frontiers = (int*) malloc(frontiersSize);
	
	// allocate device memory
	printf("Allocate device memory memory ...\n");
	hipMalloc((void **) &d_vertices, verticesSize);
	hipMalloc((void **) &d_edges, edgesSize);
	hipMalloc((void **) &d_depths, depthsSize);
	hipMalloc((void **) &d_frontiers, frontiersSize);

	// copy from file to host memory
	int n = 0;
	int counter = 0;
	int edgeCounter = 0;
	int startVertex, endVertex;

	printf("Reading file ...\n");
	for (int i=0; i<nVertices; i++) {
		h_vertices[i] = edgeCounter;
		fscanf(fp, "%d", &n);

		for (int j=0; j<n; j++) {
			fscanf(fp, "%d", &startVertex);
			fscanf(fp, "%d", &endVertex);	
			h_edges[edgeCounter] = endVertex;
			edgeCounter++;
		}
	}
    fclose(fp);
	// Set the last vertice (which has index nVertices + 1) with the appropriate value. 
	// This vertice exists only for calculation purposes.
	h_vertices[nVertices] = edgeCounter;

	// init depths, frontiers and mask
	for (int i=0; i<nVertices; i++) {
		h_depths[i] = -1;
		h_frontiers[i] = 0;
	}

	// print check
	/*
	for (int i=0; i<nVertices+1; i++) {
		cout << h_vertices[i] << " ";
	}
	cout << "\n";

	for (int i=0; i<nEdges; i++) {
		cout << h_edges[i] << " ";
	}
	cout << "\n";
	*/

	// set the starting vertex as frontier
	h_frontiers[0] = 1; 
	// set the root vertex depth = 0
	h_depths[0] = 0;

	// init gpu arrays
	printf("Init device memory ...\n");
	hipMemset(d_vertices, 0, verticesSize);
	hipMemset(d_edges, 0, edgesSize);
	hipMemset(d_depths, -1, depthsSize);
	hipMemset(d_frontiers, 0, frontiersSize);

	// copy to gpu
	printf("Copy memory to device ...\n");
	hipMemcpy(d_vertices, h_vertices, verticesSize, hipMemcpyHostToDevice);
	hipMemcpy(d_edges, h_edges, edgesSize, hipMemcpyHostToDevice);
	hipMemcpy(d_depths, h_depths, depthsSize, hipMemcpyHostToDevice);
	hipMemcpy(d_frontiers, h_frontiers, frontiersSize, hipMemcpyHostToDevice);

	int h_hasChanges = 1;
	int nBlocks = nVertices / NUMBER_OF_THREADS_PER_BLOCK;
	if (nVertices % NUMBER_OF_THREADS_PER_BLOCK) {
		nBlocks++;
	}
	printf("\nNumber of blocks: %d\n", nBlocks);
	printf("Number of threads per blocks: %d\n", NUMBER_OF_THREADS_PER_BLOCK);
	printf("\nKernel launched\n");
	timer.Start();
	while (h_hasChanges) {
		// reset the flag to 'false' and copy the value to the device
		h_hasChanges = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(d_hasChanges), &d_hasChanges, sizeof(int), 0, hipMemcpyHostToDevice);

		// kernel
		bfs<<<nBlocks, NUMBER_OF_THREADS_PER_BLOCK>>>(d_vertices, d_edges, d_depths, d_frontiers, nVertices);

		// get the flag from the device
		hipMemcpyFromSymbol(&h_hasChanges, HIP_SYMBOL(d_hasChanges), sizeof(int), 0, hipMemcpyDeviceToHost);
		//printf("MAIN LOOP FINISHED\n");
	}
	timer.Stop();
	
	cout << "Total time: " << timer.Elapsed() << "ms\n";

	// copy to host
	hipMemcpy(h_depths, d_depths, depthsSize, hipMemcpyDeviceToHost);

	// write to file
	fp1 = fopen("D:\\output.txt", "w+");
	for (int i=0; i<nVertices; i++) {
		fprintf(fp1, "%d:\t%d\n", i, h_depths[i]);
	}
	fclose(fp1);


	// print 
	/*
	for (int i=0; i<nVertices; i++) {
		cout << h_depths[i] << " ";
	}
	*/

	// wait
	scanf("%d", NULL);
    
	return 0;
}

