#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include <stdio.h>
#include <stdlib.h> 
#include <iostream>
using namespace std;

const int matrixSize = 4;

__global__ void matrix_add_mul(int *a, int *b, int *c, int * d, int width)
{
	int threadId = threadIdx.x + threadIdx.y * blockDim.x;
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int globalId = blockId * blockDim.x * blockDim.y + threadId;
	/*int absX = threadIdx.x + blockDim.x * blockIdx.x;
	int absY = threadIdx.y + blockDim.y * blockIdx.y;*/
	//int globalId = absX + absY * width;
	int absX = globalId % width;
	int absY = globalId / width;

    c[globalId] = a[globalId] + b[globalId];
	//printf("%d ", blockIdx.x);

	for (int i=0; i<width; i++) 
	{
		int sum = a[absY * width + i] * b[absX + i*width];
		d[globalId] += sum;
		
	}


}

int main()
{
	int *h_a, *h_b, *h_c, *h_d;	
	int *d_a, *d_b, *d_c, *d_d;
	int bytes = matrixSize * matrixSize * sizeof(int);
	GpuTimer timer;

	h_a = (int*) malloc(bytes);
	h_b = (int*) malloc(bytes);
	h_c = (int*) malloc(bytes);
	h_d = (int*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);
	hipMalloc((void **) &d_c, bytes);
	hipMalloc((void **) &d_d, bytes);

	// init host arrays
	for (int i=0; i<matrixSize*matrixSize; i++) 
	{
			h_a[i] = i+1;
			h_b[i] = i+1;
			h_c[i] = 0;
			h_d[i] = 0;
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);
	hipMemset(d_c, 0, bytes);
	hipMemset(d_d, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
	
	
	// kernel call
	timer.Start();
	matrix_add_mul<<<4, 4>>>(d_a, d_b, d_c, d_d, matrixSize);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	// copy to host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
	for (int i=0; i<matrixSize; i++)
	{
		for (int j=0; j<matrixSize; j++)
		{
		cout << " " << h_c[i*matrixSize + j];
		}
		cout << endl;
	}

	cout << endl << endl;

	hipMemcpy(h_d, d_d, bytes, hipMemcpyDeviceToHost);
	for (int i=0; i<matrixSize; i++)
	{
		for (int j=0; j<matrixSize; j++)
		{
		cout << " " << h_d[i*matrixSize + j];
		}
		cout << endl;
	}
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_d);
	scanf("%d", NULL);
    return 0;
}

