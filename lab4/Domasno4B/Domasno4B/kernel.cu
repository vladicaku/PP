#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include <stdio.h>
#include <stdlib.h> 
#include <iostream>
using namespace std;

const int arraySize = 20480;

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ int getGlobalIdx_1D_1D() 
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void adj_diff_global(float *a, float *b, int len)
{
	int globalId = getGlobalIdx_1D_1D();

	if (globalId < len - 1) 
	{
		b[globalId] = a[globalId+1]-a[globalId];
	}
	else
	{
		b[globalId] = a[globalId];
	}
}

__global__ void adj_diff_global_1(float *a, int len)
{
	int globalId = getGlobalIdx_1D_1D();
	float next, curr;

	if (globalId < len - 1) 
	{
		curr = a[globalId];
		next = a[globalId+1];
		__syncthreads();
		a[globalId] = next-curr;
	}
	else
	{
		curr = a[globalId];
		__syncthreads();
		a[globalId] = curr;
	}
}

__global__ void adj_diff_global_2(float *a, float *b, int len)
{
	int globalId = getGlobalIdx_1D_1D();
	b[globalId] = a[globalId+1]-a[globalId];

	globalId += len / 2;
	if (globalId < len - 1) 
	{
		b[globalId] = a[globalId+1]-a[globalId];
	}
	else
	{
		b[globalId] = a[globalId];
	}
}

__global__ void adj_diff_shared(float *a, float *b, int len)
{
	extern __shared__ float niza[];
	int globalId = getGlobalIdx_1D_1D();
	int inBlockId = threadIdx.x + threadIdx.y * blockDim.x;
	float curr, next;

	niza[inBlockId] = a[globalId];

	if (inBlockId == blockDim.x - 1) {
		niza[inBlockId+1] = a[globalId+1];
	}
	__syncthreads();

	curr = niza[inBlockId];
	next = niza[inBlockId+1];
	__syncthreads();

	if (globalId < len - 1) 
	{	
		niza[inBlockId] = next-curr;
	}
	else
	{	
		niza[inBlockId] = curr;
	}

	b[globalId] = niza[inBlockId];
}

void test1()
{
	float *h_a, *h_b;
	float *d_a, *d_b;
	int bytes = arraySize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);
	h_b = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);

	// init host arrays
	for (int i=0; i<arraySize; i++) 
	{
		h_a[i] = i;
		h_b[i] =0;
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);


	// kernel call
	timer.Start();
	adj_diff_global<<<20, 1024>>>(d_a, d_b, arraySize);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	timer.Start();
	adj_diff_global<<<40, 512>>>(d_a, d_b, arraySize);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	timer.Start();
	adj_diff_global<<<80, 256>>>(d_a, d_b, arraySize);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	timer.Start();
	adj_diff_global<<<160, 128>>>(d_a, d_b, arraySize);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	timer.Start();
	adj_diff_global<<<640, 32>>>(d_a, d_b, arraySize);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	// copy to host
	/*hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
	for (int i=0; i<arraySize; i++)
	{
		cout << " " << h_b[i];
	}	
	cout << "\n";*/
}

void test2(int g, int b)
{
	float *h_a;
	float *d_a;
	int bytes = arraySize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);

	// init host arrays
	for (int i=0; i<arraySize; i++) 
	{
		h_a[i] = i;
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);


	// kernel call
	timer.Start();
	adj_diff_global_1<<<g, b>>>(d_a, arraySize);
	timer.Stop();
	cout << timer.Elapsed() << "\t<<<" << g << ", " << b << ">>> " << "\n";

	// copy to host
	/*hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
	for (int i=0; i<arraySize; i++)
	{
		cout << " " << h_a[i];
	}
	cout << "\n";*/
	
}

void test3(int g, int b)
{
	float *h_a, *h_b;
	float *d_a, *d_b;
	int bytes = arraySize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);
	h_b = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);

	// init host arrays
	for (int i=0; i<arraySize; i++) 
	{
		h_a[i] = i;
		h_b[i] = 0;
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);


	// kernel call
	timer.Start();
	adj_diff_shared<<<g, b, b*sizeof(float) + sizeof(float)>>>(d_a, d_b, arraySize);
	timer.Stop();
	cout << timer.Elapsed() << "\t<<<" << g << ", " << b << ">>> " << "\n";

	// copy to host
	/*hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
	for (int i=0; i<arraySize; i++)
	{
		cout << " " << h_b[i];
	}
	cout << "\n";*/
}

void test4(int g, int b)
{
	float *h_a, *h_b;
	float *d_a, *d_b;
	int bytes = arraySize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);
	h_b = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);

	// init host arrays
	for (int i=0; i<arraySize; i++) 
	{
		h_a[i] = i;
		h_b[i] = 0;
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);


	// kernel call
	timer.Start();
	adj_diff_global_2<<<g, b>>>(d_a, d_b, arraySize);
	timer.Stop();
	cout << timer.Elapsed() << "\t<<<" << g << ", " << b << ">>> " << "\n";

	// copy to host
	/*hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
	for (int i=0; i<arraySize; i++)
	{
		cout << " " << h_b[i];
	}	
	cout << "\n";*/
}

int main()
{
	// global memory (no sync);
	cout << "Global (no sync)\n----------------------------------\n";
	test1();


	// global memory (with sync);
	cout << "\nGlobal (with sync)\n----------------------------------\n";
	test2(20, 1024);
	test2(40, 512);
	test2(80, 256);
	test2(160, 128);
	test2(640, 32);


	// shared memory
	cout << "\nShared\n----------------------------------\n";
	test3(20, 1024);
	test3(40, 512);
	test3(80, 256);
	test3(160, 128);
	test3(640, 32);
	

	// poveke rabota
	cout << "\nMore work\n----------------------------------\n";
	test4(10, 1024);
	test4(20, 512);
	test4(40, 256);
	test4(80, 128);
	test4(320, 32);

	scanf("%d", NULL);
	return 0;
}