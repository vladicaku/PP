#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h> 
#include <iostream>
using namespace std;

const int arraySize = 100;

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ int getGlobalIdx_1D_1D() 
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void increment_counter(float *a, int len)
{
	int globalId = getGlobalIdx_2D_2D();
	int id = globalId % len;
	atomicAdd(&a[id], 1);
}

void test1(int g, int b)
{
	float *h_a;
	float *d_a;
	int bytes = arraySize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);

	// init host arrays
	for (int i=0; i<arraySize; i++) 
	{
		h_a[i] = 0;
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);


	// kernel call
	timer.Start();
	increment_counter<<<g, b>>>(d_a, arraySize);
	timer.Stop();
	cout << timer.Elapsed() << "\t<<<" << g << ", " << b << ">>> " << "\n";

	// copy to host
	/*hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
	for (int i=0; i<arraySize; i++)
	{
		cout << h_a[i] << " ";
	}	
	cout << "\n";*/
}

int main()
{
	test1(1, 1024);
	test1(64, 1024);
	test1(512, 1024);
	test1(1024, 1024);
	scanf("%d", NULL);
	return 0;
}