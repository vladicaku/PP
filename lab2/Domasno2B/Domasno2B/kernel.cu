#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include <stdio.h>
#include <stdlib.h> 
#include <iostream>
using namespace std;

const int matrixSize = 1024;

__global__ void matrix_add(float *a, float *b, float *c)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int id = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	//int id = threadIdx.x + threadIdx.y * blockDim.x;
    c[id] = a[id] + b[id];
}

int main()
{
    float *h_a, *h_b, *h_c;	
	float *d_a, *d_b, *d_c;
	int bytes = matrixSize * matrixSize * sizeof(float);
	GpuTimer timer;

	h_a = (float*) malloc(bytes);
	h_b = (float*) malloc(bytes);
	h_c = (float*) malloc(bytes);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes);
	hipMalloc((void **) &d_c, bytes);

	// init host arrays
	for (int i=0; i<matrixSize*matrixSize; i++) 
	{
			h_a[i] = i;
			h_b[i] = i;
			h_c[i] = 0;
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);
	hipMemset(d_c, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
	
	
	// kernel call
	timer.Start();
	matrix_add<<<8192, 128>>>(d_a, d_b, d_c);
	timer.Stop();
	cout << timer.Elapsed() << "\n";

	// copy to host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
	/*for (int i=0; i<matrixSize; i++)
	{
		for (int j=0; j<matrixSize; j++)
		{
		cout << " " << h_c[i*matrixSize + j];
		}
		cout << endl;
	}*/
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	scanf("%d", NULL);
    return 0;
}
