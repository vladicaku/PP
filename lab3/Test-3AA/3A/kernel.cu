#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
using namespace std;


__global__ void rgba_to_greyscale(const uchar4* const rgbaImage, unsigned char* const greyImage, int numRows, int numCols)
{
	//threadId = threadIdx.x + threadIdx.y * blockDim.x;
}

__global__ void rgba_to_greyscalel(const uchar4* const rgbaImage, unsigned char* const greyImage, int numRows, int numCols)
{
	int abs_x = blockIdx.x * blockDim.x + threadIdx.x;
	int abs_y = blockIdx.y * blockDim.y + threadIdx.y;
	int globalId;

	if (abs_x <= numCols && abs_y <= numRows)
	{
		globalId = abs_x + abs_y * numCols;
		greyImage[globalId] = .299f * rgbaImage[globalId].x + .587f * rgbaImage[globalId].y + .114f * rgbaImage[globalId].z;
		//rgba_to_greyscale(rgbaImage, greyImage, numRows, numCols);
	}

	printf("[%d, %d] %d \n", blockIdx.x, blockIdx.y, i);/*
	if (blockIdx.x * blockDim.x > numCols)
	{
	}
	else if (blockIdx.y * blockDim.y > numRows)
	{
		
	}
	else 
	{
		int threadId = threadIdx.x + threadIdx.y * blockDim.x;
		int qusi_blockId = blockIdx.y * numCols * blockDim.y + blockIdx.x * blockDim.x * blockDim.y;
		int globalId = qusi_blockId + threadId;
		greyImage[globalId] = .299f * rgbaImage[globalId].x + .587f * rgbaImage[globalId].y + .114f * rgbaImage[globalId].z;
	}*/
}

int main()
{
	rgba_to_greyscalel<<<dim3(2,1,1), 1>>>(100);
	hipDeviceReset();
	scanf("%d", NULL);
    return 0;
}

