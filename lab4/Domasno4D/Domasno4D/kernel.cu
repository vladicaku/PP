#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
using namespace std;

const int matrixSize = 1024;
const int numberOfColors = 2;

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ int getGlobalIdx_1D_1D() 
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void histogram(unsigned int *a, unsigned int *b)
{
	int globalId = getGlobalIdx_1D_1D();
	atomicAdd(&b[a[globalId]], 1);
}

void test1(int g, int b)
{
	unsigned int *h_a, *d_a;
	unsigned int *h_b, *d_b;
	unsigned int bytes = matrixSize * matrixSize * sizeof(unsigned int);
	unsigned int bytes1 = numberOfColors * sizeof(unsigned int);
	GpuTimer timer;

	h_a = (unsigned int*) malloc(bytes);
	h_b = (unsigned int*) malloc(bytes1);

	hipMalloc((void **) &d_a, bytes);
	hipMalloc((void **) &d_b, bytes1);

	// init host arrays
	srand(time(NULL));
	for (int i=0; i<matrixSize; i++) 
	{
		for (int j=0; j<matrixSize; j++)
		{
			h_a[i*matrixSize + j] = rand() % numberOfColors;
		}
	}

	for (int i=0; i<numberOfColors; i++)
	{
		h_b[i] = 0;
	}

	// init gpu arrays
	hipMemset(d_a, 0, bytes);
	hipMemset(d_b, 0, bytes);

	// copy to gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes1, hipMemcpyHostToDevice);


	// kernel call
	timer.Start();
	histogram<<<g, b>>>(d_a, d_b);
	timer.Stop();
	cout << timer.Elapsed() << "\t<<<" << g << ", " << b << ">>> " << "\n";

	// copy to host
	/*hipMemcpy(h_b, d_b, bytes1, hipMemcpyDeviceToHost);
	for (int i=0; i<numberOfColors; i++)
	{
		cout << h_b[i] << " ";
	}	
	cout << "\n";*/
}

int main()
{
	test1(1024, 1024);
	scanf("%d", NULL);
	return 0;
}